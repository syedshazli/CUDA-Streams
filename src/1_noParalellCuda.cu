#include "hip/hip_runtime.h"
// kernels are run sequentially
#define N 1024;
__global__ void add(int *A, int *B, int *C){

	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	C[tid] = B[tid] + A[tid];

}


__global__ void multiply(int *A, int *B, int *C){

	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	C[tid] = B[tid] * A[tid];
}


int main(){

	int *dev_add_a;
	int *dev_add_b;
	int *dev_add_c;
	
	int add_a[N];
	int add_b[N];
	int add_c[N];
	
	int *dev_mult_a;
	int *dev_mult_b;
	int *dev_mult_c;
	
	int mult_a[N];
	int mult_b[N];
	int mult_c[N];


	// to benchmark/compare, if we want to run concurrently, all vars must be initialised

	hipMalloc((void**)&dev_add_a, sizeof(int) * N);
	hipMalloc((void**)&dev_add_b, sizeof(int) * N);
	hipMalloc((void**)&dev_add_c, sizeof(int) * N);
	
	for(int i = 0; i< N; i++){
		
		add_a[i] = i;
		add_b[i] = i-1;

		mult_a[i] = i;
		mult_b[i] = i-1;
	
	}

	// if we're using the results of some array and not just storing, a memcpy from host to device is needed
	// assumes arrays a and b are pre filled
	cudaMemCpy();


}

